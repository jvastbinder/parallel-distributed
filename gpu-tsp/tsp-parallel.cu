#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <limits.h>
#include <time.h>

// CUDA runtime
#include "/usr/local/cuda-9.0/include/hip/hip_runtime.h"

/* Original permuation code due to D. Jimenez, UT Austin
 * http://faculty.cse.tamu.edu/djimenez/ut/utsa/cs3343/
 */

/* Requires C99 compiler (gcc: -std=c99) */
#define DEBUG 0
#define debug_printf(fmt, ...) do { if (DEBUG) fprintf(stderr, fmt, __VA_ARGS__); } while (0)

/* Reference an element in the TSP distance array. */
#define TSP_ELT(tsp, n, i, j) *(tsp + (i * n) + j)

/* Trivial action to pass to permutations--print out each one. */
void
print_perm(int *perm, int n, char *msge)
{
  for (int j = 0;  j < n;  j++) {
	printf("%2d ", perm[j]);
  }
  printf(" - %s\n", msge);
}

/* No-op action */
void
nop(int *v, int n)
{
  return;
}

/* Create an instance of a symmetric TSP. */
int *
create_tsp(int n, int random_seed)
{
  int *tsp = (int *) malloc(n * n * sizeof(int));

  srandom(random_seed);
  for (int i = 0;  i < n;  i++) {
	for (int j = 0;  j <= i;  j++) {
	  int val = (int)(random() / (RAND_MAX / 100));
	  TSP_ELT(tsp, n, i, j) = val;
	  TSP_ELT(tsp, n, j, i) = val;
	}
  }
  return tsp;
}

/* Print a TSP distance matrix. */
void
print_tsp(int *tsp, int n, int random_seed)
{
  printf("TSP (%d cities - seed %d)\n    ", n, random_seed);
  for (int j = 0;  j < n;  j++) {
	printf("%3d|", j);
  }
  printf("\n");
  for (int i = 0;  i < n;  i++) {
	printf("%2d|", i);
	for (int j = 0;  j < n;  j++) {
	  printf("%4d", TSP_ELT(tsp, n, i, j));
	}
	printf("\n");
  }
  printf("\n");
}

__device__ void
print_tsp_d(int *tsp, int n, int random_seed)
{
  printf("TSP (%d cities - seed %d)\n    ", n, random_seed);
  for (int j = 0;  j < n;  j++) {
	printf("%3d|", j);
  }
  printf("\n");
  for (int i = 0;  i < n;  i++) {
	printf("%2d|", i);
	for (int j = 0;  j < n;  j++) {
	  printf("%4d", TSP_ELT(tsp, n, i, j));
	}
	printf("\n");
  }
  printf("\n");
}

void
usage(char *prog_name)
{
  fprintf(stderr, "usage: %s [flags]\n", prog_name);
  fprintf(stderr, "   -h\n");
  fprintf(stderr, "   -t <number of threads>\n");
  fprintf(stderr, "   -c <number of cities>\n");
  fprintf(stderr, "   -s <random seed>\n");
  exit(1);
}

__device__ long
factorial(int n)
{
    if (n < 1) {
        return 0;
    }

    long rtn = 1;
    for (int i = 1;  i <= n;  i++) {
        rtn *= i;
    }
    return rtn;
}


__device__ int 
calc_cost(int *perm, int *matrix, int n)
{
    /* Calculate the length of the tour for the current permutation. */
    int total = 0;
    for (int i = 0;  i < n;  i++) {
        int j = (i + 1) % n;
        int from = perm[i];
        int to = perm[j];
        int val = TSP_ELT(matrix, n, from, to);
        total += val;
    }

    return total;
}

void 
create_tsp(int *matrix, int n, int random_seed)
{
    srandom(random_seed);
    for (int i = 0;  i < n;  i++) {
        for (int j = 0;  j <= i;  j++) {
            int val = (int)(random() / (RAND_MAX / 100));
            TSP_ELT(matrix, n, i, j) = val;
            TSP_ELT(matrix, n, j, i) = val;
        }
    }
}

/**** List ADT ****************/

typedef struct {
  int *values;					/* Values stored in list */
  int max_size;					/* Maximum size allocated */
  int cur_size;					/* Size currently in use */
} list_t;

/* Dump list, including sizes */
__device__ void
list_dump(list_t *list)
{
  printf("%2d/%2d", list->cur_size, list->max_size);
  for (int i = 0;  i < list->cur_size;  i++) {
	printf(" %d", list->values[i]);
  }
  printf("\n");
}

/* Allocate list that can store up to 'max_size' elements */
__device__ list_t *
list_alloc(int max_size)
{
  list_t *list = (list_t *)malloc(sizeof(list_t));
  list->values = (int *)malloc(max_size * sizeof(int));
  list->max_size = max_size;
  list->cur_size = 0;
  return list;
}

/* Free a list; call this to avoid leaking memory! */
__device__ void
list_free(list_t *list)
{
  free(list->values);
  free(list);
}

/* Add a value to the end of the list */
__device__ void
list_add(list_t *list, int value)
{
  if (list->cur_size >= list->max_size) {
	printf("List full");
	list_dump(list);
  }
  list->values[list->cur_size++] = value;
}

/* Return the current size of the list */
__device__ int
list_size(list_t *list)
{
  return list->cur_size;
}

/* Validate index */
__device__ void
_list_check_index(list_t *list, int index)
{
  if (index < 0 || index > list->cur_size - 1) {
	printf("Invalid index %d\n", index);
	list_dump(list);
  }
}

/* Get the value at given index */
__device__ int
list_get(list_t *list, int index)
{
  _list_check_index(list, index);
  return list->values[index];
}

/* Remove the value at the given index */
__device__ void
list_remove_at(list_t *list, int index)
{
  _list_check_index(list, index);
  for (int i = index; i < list->cur_size - 1;  i++) {
	list->values[i] = list->values[i + 1];
  }
  list->cur_size--;
}

/* Retrieve a copy of the values as a simple array of integers. The returned
   array is allocated dynamically; the caller must free the space when no
   longer needed.
 */
__device__ int *
list_as_array(list_t *list)
{
  int *rtn = (int *)malloc(list->max_size * sizeof(int));
  for (int i = 0;  i < list->max_size;  i++) {
	rtn[i] = list_get(list, i);
  }
  return rtn;
}

/**** Permutation ****************/

/* Permutation algorithms based on code found at:
   http://www.mathblog.dk/project-euler-24-millionth-lexicographic-permutation/
   which references:
   http://www.cut-the-knot.org/do_you_know/AllPerm.shtml
*/

/* Return the kth lexographically ordered permuation of an array of k integers
   in the range [0 .. size - 1]. The integers are allocated dynamically and
   should be free'd by the caller when no longer needed.
*/
__device__ int *
kth_perm(int k, int size)
{
  long remain = k - 1;

  list_t *numbers = list_alloc(size);
  for (int i = 0;  i < size;  i++) {
	list_add(numbers, i);
  }

  list_t *perm = list_alloc(size);

#if DEBUG
  printf("k=%d, size=%d, remain=%ld\n", k, size, remain);
  printf("  perm");
  list_dump(perm);
  printf("  nums");
  list_dump(numbers);
#endif

  for (int i = 1;  i < size;  i++) {
	long f = factorial(size - i);
	long j = remain / f;
	remain = remain % f;
#if DEBUG
	printf("i=%d, f=%ld j=%ld, remain=%ld\n", i, f, j, remain);
#endif

	list_add(perm, list_get(numbers, j));
	list_remove_at(numbers, j);

#if DEBUG
	printf("  perm");
	list_dump(perm);
	printf("  nums");
	list_dump(numbers);
#endif

	if (remain == 0) {
	  break;
	}
  }

  /* Append remaining digits */
  for (int i = 0;  i < list_size(numbers);  i++) {
	list_add(perm, list_get(numbers, i));
  }

  int *rtn = list_as_array(perm);
  list_free(perm);

  return rtn;
}

/* Swap v[i] and v[j] */
__device__ void
swap(int *v, int i, int j)
{
  int t = v[i];
  v[i] = v[j];
  v[j] = t;
}

/* Print a permutation array */
__device__ void
print_perm(int *perm, int size)
{
  for (int k = 0; k < size; k++) {
	printf("%4d", perm[k]);
  }
  printf("\n");
}

/* Given an array of size elements at perm, update the array in place to
   contain the lexographically next permutation. It is originally due to
   Dijkstra. The present version is discussed at:
   http://www.cut-the-knot.org/do_you_know/AllPerm.shtml
 */
__device__ void
next_perm(int *perm, int size)
{
  int i = size - 1;
  while (perm[i - 1] >= perm[i]) {
	i = i - 1;
  }

  int j = size;
  while (perm[j - 1] <= perm[i - 1]) {
	j = j - 1;
  }

  swap(perm, i - 1, j - 1);

  i++;
  j = size;
  while (i < j) {
	swap(perm, i - 1, j - 1);
	i++;
	j--;
  }
}


__global__ void
perm_kernel(int *glob_cost_matrix, int *min_matrix, int num_cities, int num_threads)
{
  //  extern __shared__ int cost_matrix[];
   // memcpy(cost_matrix, glob_cost_matrix, sizeof(int) * num_cities * num_cities);
    int *perm = (int *) malloc(sizeof(int) * num_cities);
    int min_cost = INT_MAX;
    int tid = threadIdx.x;
    unsigned long num_iters = factorial(num_cities) / num_threads;
    perm = kth_perm((num_iters * tid) + 1, num_cities);

    int cost;
    for(int i = 0; i < num_iters; i++)
    {
        cost = calc_cost(perm, glob_cost_matrix, num_cities);
        if(cost < min_cost)
        {
            min_cost = cost;
        }
        next_perm(perm, num_cities);
    }

    min_matrix[tid] = min_cost;
}

int
main(int argc, char **argv)
{
  int num_cities = 3;
  int random_seed = 42;
  int num_threads = 5;

  int ch;
  while ((ch = getopt(argc, argv, "t:c:hs:")) != -1) {
	switch (ch) {
	case 'c':
	  num_cities = atoi(optarg);
	  break;
	case 's':
	  random_seed = atoi(optarg);
	  break;
	case 't':
	  num_threads = atoi(optarg);
	  break;
	case 'h':
	default:
	  usage(argv[0]);
	}
  }

  int cost_matrix_size = sizeof(int) * num_cities * num_cities;
  int min_matrix_size = sizeof(int) * num_threads;

  //Initialize matrices
  int *min_matrix_h = (int *) malloc(min_matrix_size);
  int *cost_matrix_h = (int *) malloc(cost_matrix_size);
  int *min_matrix_d, *cost_matrix_d;
  hipMalloc((void **) &min_matrix_d, min_matrix_size);
  hipMalloc((void **) &cost_matrix_d, cost_matrix_size);

  //create and copy cost matrix
  create_tsp(cost_matrix_h, num_cities, random_seed);
  print_tsp(cost_matrix_h, num_cities, random_seed);
  hipMemcpy(cost_matrix_d, cost_matrix_h, cost_matrix_size, hipMemcpyHostToDevice);

  dim3 blocks_per_grid(1);
  dim3 threads_per_block(num_threads);


  perm_kernel<<<blocks_per_grid, threads_per_block>>> (cost_matrix_d, min_matrix_d, num_cities, num_threads);

  hipMemcpy(min_matrix_h, min_matrix_d, min_matrix_size, hipMemcpyDeviceToHost);

  int shortest_length = INT_MAX;
  for(int i = 0; i < num_threads; i++){
      printf("%d ",min_matrix_h[i]);
      if(min_matrix_h[i] < shortest_length)
      {
          shortest_length = min_matrix_h[i];
      }
  }
  printf("\n");
  printf("\n");
  printf("Shortest %d", shortest_length);
  printf("\n");
}

