
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <limits.h>
#include <time.h>

/* Original permuation code due to D. Jimenez, UT Austin
 * http://faculty.cse.tamu.edu/djimenez/ut/utsa/cs3343/
 */

/* Reference an element in the TSP distance array. */
#define TSP_ELT(tsp, n, i, j) *(tsp + (i * n) + j)

/* Create an instance of a symmetric TSP. */
int *
create_tsp(int n, int random_seed)
{
  int *tsp = (int *) malloc(n * n * sizeof(int));

  srandom(random_seed);
  for (int i = 0;  i < n;  i++) {
	for (int j = 0;  j <= i;  j++) {
	  int val = (int)(random() / (RAND_MAX / 100));
	  TSP_ELT(tsp, n, i, j) = val;
	  TSP_ELT(tsp, n, j, i) = val;
	}
  }
  return tsp;
}

/* Print a TSP distance matrix. */
void
print_tsp(int *tsp, int n, int random_seed)
{
  printf("TSP (%d cities - seed %d)\n    ", n, random_seed);
  for (int j = 0;  j < n;  j++) {
	printf("%3d|", j);
  }
  printf("\n");
  for (int i = 0;  i < n;  i++) {
	printf("%2d|", i);
	for (int j = 0;  j < n;  j++) {
	  printf("%4d", TSP_ELT(tsp, n, i, j));
	}
	printf("\n");
  }
  printf("\n");
}

void
usage(char *prog_name)
{
  fprintf(stderr, "usage: %s [flags]\n", prog_name);
  fprintf(stderr, "   -h\n");
  fprintf(stderr, "   -t <number of threads>\n");
  fprintf(stderr, "   -c <number of cities>\n");
  fprintf(stderr, "   -s <random seed>\n");
  exit(1);
}

__device__ unsigned long
factorial(int n)
{
    if (n < 1) {
        return 0;
    }

    unsigned long rtn = 1;
    for (unsigned i = 1;  i <= n;  i++) {
        rtn *= i;
    }
    return rtn;
}


__device__ int 
calc_cost(int *perm, int *matrix, int n)
{
    int total = 0;
    for (int i = 0;  i < n;  i++) {
        int j = (i + 1) % n;
        int from = perm[i];
        int to = perm[j];
        int val = TSP_ELT(matrix, n, from, to);
        total += val;
    }

    return total;
}

void 
create_tsp(int *matrix, int n, int random_seed)
{
    srandom(random_seed);
    for (int i = 0;  i < n;  i++) {
        for (int j = 0;  j <= i;  j++) {
            int val = (int)(random() / (RAND_MAX / 100));
            TSP_ELT(matrix, n, i, j) = val;
            TSP_ELT(matrix, n, j, i) = val;
        }
    }
}

/**** List ADT ****************/

typedef struct {
  int *values;					/* Values stored in list */
  int max_size;					/* Maximum size allocated */
  int cur_size;					/* Size currently in use */
} list_t;

/* Dump list, including sizes */
__device__ void
list_dump(list_t *list)
{
  printf("%2d/%2d", list->cur_size, list->max_size);
  for (int i = 0;  i < list->cur_size;  i++) {
	printf(" %d", list->values[i]);
  }
  printf("\n");
}

/* Allocate list that can store up to 'max_size' elements */
__device__ list_t *
list_alloc(int max_size)
{
  list_t *list = (list_t *)malloc(sizeof(list_t));
  list->values = (int *)malloc(max_size * sizeof(int));
  list->max_size = max_size;
  list->cur_size = 0;
  return list;
}

/* Free a list; call this to avoid leaking memory! */
__device__ void
list_free(list_t *list)
{
  free(list->values);
  free(list);
}

/* Add a value to the end of the list */
__device__ void
list_add(list_t *list, int value)
{
  if (list->cur_size >= list->max_size) {
	printf("List full");
	list_dump(list);
  }
  list->values[list->cur_size++] = value;
}

/* Return the current size of the list */
__device__ int
list_size(list_t *list)
{
  return list->cur_size;
}

/* Validate index */
__device__ void
_list_check_index(list_t *list, unsigned long index)
{
  if (index > list->cur_size - 1) {
	printf("Invalid index %d\n", index);
	list_dump(list);
  }
}

/* Get the value at given index */
__device__ int
list_get(list_t *list, unsigned long index)
{
  _list_check_index(list, index);
  return list->values[index];
}

/* Remove the value at the given index */
__device__ void
list_remove_at(list_t *list, int index)
{
  _list_check_index(list, index);
  for (int i = index; i < list->cur_size - 1;  i++) {
	list->values[i] = list->values[i + 1];
  }
  list->cur_size--;
}

/* Retrieve a copy of the values as a simple array of integers. The returned
   array is allocated dynamically; the caller must free the space when no
   longer needed.
 */
__device__ int *
list_as_array(list_t *list)
{
  int *rtn = (int *)malloc(list->max_size * sizeof(int));
  for (int i = 0;  i < list->max_size;  i++) {
	rtn[i] = list_get(list, i);
  }
  return rtn;
}

/**** Permutation ****************/

/* Permutation algorithms based on code found at:
   http://www.mathblog.dk/project-euler-24-millionth-lexicographic-permutation/
   which references:
   http://www.cut-the-knot.org/do_you_know/AllPerm.shtml
*/

/* Return the kth lexographically ordered permuation of an array of k integers
   in the range [0 .. size - 1]. The integers are allocated dynamically and
   should be free'd by the caller when no longer needed.
*/
__device__ int *
kth_perm(int k, int size)
{
  long remain = k - 1;

  list_t *numbers = list_alloc(size);
  for (int i = 0;  i < size;  i++) {
	list_add(numbers, i);
  }

  list_t *perm = list_alloc(size);

  for (int i = 1;  i < size;  i++) {
	unsigned long f = factorial(size - i);
	unsigned long j = remain / f;
	remain = remain % f;

	list_add(perm, list_get(numbers, j));
	list_remove_at(numbers, j);

	if (remain == 0) {
	  break;
	}
  }

  /* Append remaining digits */
  for (int i = 0;  i < list_size(numbers);  i++) {
	list_add(perm, list_get(numbers, i));
  }

  int *rtn = list_as_array(perm);
  list_free(perm);

  return rtn;
}

/* Swap v[i] and v[j] */
__device__ void
swap(int *v, int i, int j)
{
  int t = v[i];
  v[i] = v[j];
  v[j] = t;
}

/* Print a permutation array */
__device__ void
print_perm(int *perm, int size)
{
  for (int k = 0; k < size; k++) {
	printf("%4d", perm[k]);
  }
  printf("\n");
}

/* Given an array of size elements at perm, update the array in place to
   contain the lexographically next permutation. It is originally due to
   Dijkstra. The present version is discussed at:
   http://www.cut-the-knot.org/do_you_know/AllPerm.shtml
 */
__device__ void
next_perm(int *perm, int size)
{
  int i = size - 1;
  while (perm[i - 1] >= perm[i]) {
	i = i - 1;
  }

  int j = size;
  while (perm[j - 1] <= perm[i - 1]) {
	j = j - 1;
  }

  swap(perm, i - 1, j - 1);

  i++;
  j = size;
  while (i < j) {
	swap(perm, i - 1, j - 1);
	i++;
	j--;
  }
}


__global__ void
perm_kernel(int *glob_cost_matrix, int *min_matrix, int num_cities, int num_threads)
{
    int block_id = 
        blockIdx.x +
        blockIdx.y * blockDim.x +
        blockIdx.z * blockDim.x * blockDim.y;
    int block_offset =
        block_id *
        blockDim.x * blockDim.y * blockDim.z;
    int thread_offset =
        threadIdx.x +
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;
    int tid = block_offset + thread_offset;
    __shared__ int cost_matrix[144];
    int init_iters = (num_cities * num_cities) / num_threads;
    if(tid == 0)
        init_iters += (num_cities * num_cities) % num_threads;
    for(int i = 0; i < num_cities * num_cities; i++)
        cost_matrix[i] = glob_cost_matrix[i];
    __syncthreads();
    unsigned long num_iters = factorial(num_cities) / num_threads;
    int *perm = kth_perm((num_iters * tid) + 1, num_cities);

    int min_cost = INT_MAX;
    int cost;
    for(unsigned long i = 0; i < num_iters; i++)
    {
        cost = calc_cost(perm, cost_matrix, num_cities);
        if(cost < min_cost)
        {
            min_cost = cost;
        }
        next_perm(perm, num_cities);
    }

    min_matrix[tid] = min_cost;
}

double
now(void)
{
    struct timespec current_time;
    double ONE_BILLION = (double)1000000000.0;

    clock_gettime(CLOCK_REALTIME, &current_time);
    return current_time.tv_sec + (current_time.tv_nsec / ONE_BILLION);
}

int
main(int argc, char **argv)
{
  int num_cities = 3;
  int random_seed = 42;
  int num_threads = 5;

  int ch;
  while ((ch = getopt(argc, argv, "t:c:hs:")) != -1) {
	switch (ch) {
	case 'c':
	  num_cities = atoi(optarg);
	  break;
	case 's':
	  random_seed = atoi(optarg);
	  break;
	case 't':
	  num_threads = atoi(optarg);
	  break;
	case 'h':
	default:
	  usage(argv[0]);
	}
  }
  double start = now();

  int cost_matrix_size = sizeof(int) * num_cities * num_cities;
  int min_matrix_size = sizeof(int) * num_threads;

  //Initialize matrices
  int *min_matrix_h = (int *) malloc(min_matrix_size);
  int *cost_matrix_h = (int *) malloc(cost_matrix_size);
  int *min_matrix_d, *cost_matrix_d;
  hipMalloc(&min_matrix_d, min_matrix_size);
  hipMalloc(&cost_matrix_d, cost_matrix_size);

  //create and copy cost matrix to device
  create_tsp(cost_matrix_h, num_cities, random_seed);
//  print_tsp(cost_matrix_h, num_cities, random_seed);
  hipMemcpy(cost_matrix_d, cost_matrix_h, cost_matrix_size, hipMemcpyHostToDevice);

  //launch kernel
  int threads_per_block = num_threads;
  for(int i = 1; i < 1025; i *= 2)
  {
      if(i > num_threads)
          break;
      if(i < num_threads && (num_threads % i == 0))
      {
          threads_per_block = i;
      }
  }
  int blocks_per_grid = num_threads / threads_per_block;

  perm_kernel<<<blocks_per_grid, threads_per_block>>>(cost_matrix_d, min_matrix_d, num_cities, num_threads);

  //copy local mins back to host
  hipError_t rtn = hipMemcpy(min_matrix_h, min_matrix_d, min_matrix_size, hipMemcpyDeviceToHost);
  if(rtn != 0){
      printf("Ouchie:\n%s\n", hipGetErrorString(rtn));
  }
    

  //calculate minimum
  int shortest_length = INT_MAX;
  for(int i = 0; i < num_threads; i++){
      if(min_matrix_h[i] < shortest_length)
      {
          shortest_length = min_matrix_h[i];
      }
  }

  double stop = now();
  printf("Shortest %d\n", shortest_length);
  printf("Blocks per grid:%d\n", blocks_per_grid);
  printf("Threads per block:%d\n", threads_per_block);
  printf("Num cities:%d\n", num_cities);
  printf("Num threads:%d\n", num_threads);
  printf("Took %5.3f seconds\n\n", stop - start);
}

